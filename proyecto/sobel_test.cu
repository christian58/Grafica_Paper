#include "hip/hip_runtime.h"
//nvcc mat.cu  -o m `pkg-config opencv --cflags --libs`; ./m
#include <opencv2/highgui/highgui.hpp>
#include <bits/stdc++.h>
using namespace cv;
using namespace std;
#define THREADS_PER_BLOCK 1024//1024


//=======================CUDA================================
__global__ void cal_sobel(int *a,int *sal, int tam) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if( index < tam){
        //int val = a[index] + a[index] ;
		int t = (a[index] + b[index] + c[index])/3;
		a[index] = t; 
		b[index] = t;
		c[index] = t;
        sal[index] = val;
	}
}


void CUDA_process_img(int *A){

	int *d_A;
    int *sal;
	int nElem = rows * cols;
	int size = nElem * sizeof(int);
	//Allocate device memory for matrices
	hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &sal, size);
	//hipMalloc((void **) &d_B, size);
	//hipMalloc((void **) &d_C, size);
	//Copy B and C to device memory
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	//hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	//hipMemcpy(d_C, C, size, hipMemcpyHostToDevice);
	//run
	cal_sobel<<<(nElem+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_A,sal, nElem);//run
	
	hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost);
	//hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
	//hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	//Free device matrices
	//hipFree(d_B);
	hipFree(d_A);
	//hipFree(d_C);
}

//======================OPEN-CV=================================
//brillo en serial
void CPU_add_brillo(int *R,int *G,int *B, int value, int rows, int cols){
    for (int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            R[cols * i + j] += value;
            G[cols * i + j] += value;
            B[cols * i + j] += value;
        }
    }
}

Mat sobel_cuda(Mat &image){
    int rows = image.rows;
    int cols = image.cols;
    int nElem = rows * cols;
    int * B = (int *) malloc(nElem * sizeof(int));
    //int * G = (int *) malloc(nElem * sizeof(int));
    //int * B = (int *) malloc(nElem * sizeof(int));

    //load IMG
    for( int i = 0; i < image.rows; i++ ){
        for( int j = 0; j < image.cols; j++ ){
        	int r = image.at<Vec3b>(i,j)[0];
        	//int g = image.at<Vec3b>(i,j)[1];
        	//int b = image.at<Vec3b>(i,j)[2];
            R[cols*i+j] = r;
            //G[cols*i+j] = g;
            //B[cols*i+j] = b;

        }
    }

    //CUDA_process_img(R,G,B,value,rows,cols);

    CUDA_process_img(R);
    
    //CPU_add_brillo(R,G,B,value,rows,cols);

    Mat new_image = Mat::zeros( image.size(), image.type() );
    for( int i = 0; i < image.rows; i++ ){
        for( int j = 0; j < image.cols; j++ ){
            new_image.at<Vec3b>(i,j)[0] = saturate_cast<uchar>( R[cols*i+j] );
            //new_image.at<Vec3b>(i,j)[1] = saturate_cast<uchar>( G[cols*i+j] );
            //new_image.at<Vec3b>(i,j)[2] = saturate_cast<uchar>( B[cols*i+j] );
        }
    }
    free(R);free(G);free(B);
    return new_image;
}

void metodo_sobel(){


    Mat img = imread("lena.bmp", CV_LOAD_IMAGE_COLOR);
    imshow( "Imagen Original", img ); 
    Mat img_gray;
    GaussianBlur( img, img, Size(3,3), 0, 0, BORDER_DEFAULT );
    cvtColor( img, img_gray, COLOR_BGR2GRAY );

    Mat img2 =sobel_cuda(img_gray);
    imshow( "brillo", img2 ); 
    waitKey(0);

    /********************/
    
    //Mat img_oring, img_gray, img_sal;
    //img_oring = imread( argv[1], IMREAD_COLOR ); // Load an image
    

    /**********************/
    
    double fps = 60;
    // calcular el tiempo de espera entre cada imagen a mostrar
    int delay = 1000 / fps;
    while (true){
        if(waitKey(delay) == 27) break;
    }

}


int main(){
	metodo_sobel();
	//run();
	return 0;
}
/*
nvcc mat.cu -o m -lopencv_core -lopencv_imgproc -lopencv_highgui
*/